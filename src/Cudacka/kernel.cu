#include "hip/hip_runtime.h"
// Cuckoo Cycle, a memory-hard proof-of-work by John Tromp
// Copyright (c) 2018 Jiri Vadura - photon
// This CUDA part of Theta optimized miner is covered by the FAIR MINING license 2.1.1

#include "hip/hip_runtime_api.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <xmmintrin.h>
#include <algorithm>
#include <stdio.h>
#include <stdint.h>
#include <atomic>
#include <thread>
#include <vector>
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <string.h>
#include <math.h>

// ------ OPTIONS ---------------------------------------------------
// highly experimental, not tested, but try to set to 0 on GTX 1080 Ti or any other card with 11GB RAM+
// will save 25-40 ms or so, if it doesn't crash :)
#define VRAMSMALL 1

// ------------------------------------------------------------------


#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

typedef uint8_t u8;
typedef uint16_t u16;
typedef uint32_t u32;
typedef uint64_t u64;

typedef u32 node_t;
typedef u64 nonce_t;

typedef struct uint10
{
	uint2 edges[5];
} uint10;

#ifdef VRAMSMALL
	#ifdef _WIN32
	#define DUCK_SIZE_A 129LL
	#define DUCK_SIZE_B 83LL
	#else
	#define DUCK_SIZE_A 130LL
	#define DUCK_SIZE_B 85LL
	#endif
#else
	#ifdef _WIN32
	#define DUCK_SIZE_A 130LL
	#define DUCK_SIZE_B 85LL
	#else
	#define DUCK_SIZE_A 130LL
	#define DUCK_SIZE_B 85LL
	#endif
#endif

#define DUCK_A_EDGES (DUCK_SIZE_A * 1024LL)
#define DUCK_A_EDGES_64 (DUCK_A_EDGES * 64LL)

#define DUCK_B_EDGES (DUCK_SIZE_B * 1024LL)
#define DUCK_B_EDGES_64 (DUCK_B_EDGES * 64LL)

#define EDGEBITS 29
// number of edges
#define NEDGES ((node_t)1 << EDGEBITS)
// used to mask siphash output
#define EDGEMASK (NEDGES - 1)

#define CTHREADS 1024
#define BKTMASK4K (4096-1)

__constant__ u64 recovery[42];

#define ROTL(x,b) ( ((x) << (b)) | ( (x) >> (64 - (b))) )
#define SIPROUND \
  do { \
    v0 += v1; v2 += v3; v1 = ROTL(v1,13); \
    v3 = ROTL(v3,16); v1 ^= v0; v3 ^= v2; \
    v0 = ROTL(v0,32); v2 += v1; v0 += v3; \
    v1 = ROTL(v1,17);   v3 = ROTL(v3,21); \
    v1 ^= v2; v3 ^= v0; v2 = ROTL(v2,32); \
  } while(0)


__device__  node_t dipnode(const u64 v0i, const u64 v1i, const u64 v2i, const u64 v3i, const  nonce_t nce, const  u32 uorv) {
	u64 nonce = 2 * nce + uorv;
	u64 v0 = v0i, v1 = v1i, v2 = v2i, v3 = v3i ^ nonce;
	SIPROUND; SIPROUND;
	v0 ^= nonce;
	v2 ^= 0xff;
	SIPROUND; SIPROUND; SIPROUND; SIPROUND;
	return (v0 ^ v1 ^ v2  ^ v3) & EDGEMASK;
}

__device__ ulonglong4 Pack4edges(const uint2 e1, const  uint2 e2, const  uint2 e3, const  uint2 e4)
{
	u64 r1 = (((u64)e1.y << 32) | ((u64)e1.x));
	u64 r2 = (((u64)e2.y << 32) | ((u64)e2.x));
	u64 r3 = (((u64)e3.y << 32) | ((u64)e3.x));
	u64 r4 = (((u64)e4.y << 32) | ((u64)e4.x));
	return make_ulonglong4(r1, r2, r3, r4);
}

__global__  void FluffyRecovery(const u64 v0i, const u64 v1i, const u64 v2i, const u64 v3i, ulonglong4 * buffer, int * indexes)
{
	const int gid = blockDim.x * blockIdx.x + threadIdx.x;
	const int lid = threadIdx.x;

	__shared__ u32 nonces[42];

	if (lid < 42) nonces[lid] = 0;

	__syncthreads();

	for (int i = 0; i < 1024 * 4; i++)
	{
		u64 nonce = gid * (1024 * 4) + i;

		u64 u = dipnode(v0i, v1i, v2i, v3i, nonce, 0);
		u64 v = dipnode(v0i, v1i, v2i, v3i, nonce, 1);

		u64 a = u | (v << 32);
		u64 b = v | (u << 32);

		for (int i = 0; i < 42; i++)
		{
			if ((recovery[i] == a) || (recovery[i] == b))
				nonces[i] = nonce;
		}
	}

	__syncthreads();

	if (lid < 42)
	{
		if (nonces[lid] > 0)
			indexes[lid] = nonces[lid];
	}
}

__global__  void FluffySeed2A(const u64 v0i, const u64 v1i, const u64 v2i, const u64 v3i, ulonglong4 * buffer, int * indexes)
{
	const int gid = blockDim.x * blockIdx.x + threadIdx.x;
	const int lid = threadIdx.x;

	__shared__ uint2 tmp[64][16];
	__shared__ int counters[64];

	counters[lid] = 0;

	__syncthreads();

	for (int i = 0; i < 1024 * 16; i++)
	{
		u64 nonce = gid * (1024 * 16) + i;

		uint2 hash;

		hash.x = dipnode(v0i, v1i, v2i, v3i, nonce, 0);

		int bucket = hash.x & (64 - 1);

		__syncthreads();

		int counter = min((int)atomicAdd(counters + bucket, 1), (int)15);

		hash.y = dipnode(v0i, v1i, v2i, v3i, nonce, 1);

		if (hash.x == 0 && hash.y == 0) continue;

		tmp[bucket][counter] = hash;

		__syncthreads();

		{
			int localIdx = min(16, counters[lid]);

			if (localIdx >= 8)
			{
				int newCount = (localIdx - 8);
				counters[lid] = newCount;

				{
					int cnt = min((int)atomicAdd(indexes + lid, 8), (int)(DUCK_A_EDGES_64 - 8));

					{
						buffer[(lid * DUCK_A_EDGES_64 + cnt) / 4] = Pack4edges(tmp[lid][0], tmp[lid][1], tmp[lid][2], tmp[lid][3]);
						buffer[(lid * DUCK_A_EDGES_64 + cnt + 4) / 4] = Pack4edges(tmp[lid][4], tmp[lid][5], tmp[lid][6], tmp[lid][7]);
					}
				}

				for (int t = 0; t < newCount; t++)
				{
					tmp[lid][t] = tmp[lid][t + 8];
				}

			}
		}
	}

	__syncthreads();

	{
		int localIdx = min(16, counters[lid]);

		if (localIdx >= 4)
		{
			int cnt = min((int)atomicAdd(indexes + lid, 4), (int)(DUCK_A_EDGES_64 - 4));
			buffer[(lid * DUCK_A_EDGES_64 + cnt) / 4] = Pack4edges(tmp[lid][0], tmp[lid][1], tmp[lid][2], tmp[lid][3]);
		}
		if (localIdx > 4)
		{
			int cnt = min((int)atomicAdd(indexes + lid, 4), (int)(DUCK_A_EDGES_64 - 4));
			buffer[(lid * DUCK_A_EDGES_64 + cnt) / 4] = Pack4edges(
				tmp[lid][4],
				localIdx > 5 ? tmp[lid][5] : make_uint2(0, 0),
				localIdx > 6 ? tmp[lid][6] : make_uint2(0, 0),
				localIdx > 7 ? tmp[lid][7] : make_uint2(0, 0));
		}
	}

}

#define BKTGRAN 32
__global__  void FluffySeed2B(const  uint2 * source, ulonglong4 * destination, const  int * sourceIndexes, int * destinationIndexes, int startBlock)
{
	const int gid = blockDim.x * blockIdx.x + threadIdx.x;
	const int lid = threadIdx.x;
	const int group = blockIdx.x;

	__shared__ uint2 tmp[64][16];
	__shared__ int counters[64];

	counters[lid] = 0;

	__syncthreads();

	const int offsetMem = startBlock * DUCK_A_EDGES_64;
	const int myBucket = group / BKTGRAN;
	const int microBlockNo = group % BKTGRAN;
	const int bucketEdges = min(sourceIndexes[myBucket + startBlock], (int)(DUCK_A_EDGES_64));
	const int microBlockEdgesCount = (DUCK_A_EDGES_64 / BKTGRAN);
	const int loops = (microBlockEdgesCount / 64);

	for (int i = 0; i < loops; i++)
	{
		int edgeIndex = (microBlockNo * microBlockEdgesCount) + (64 * i) + lid;

		if (edgeIndex < bucketEdges)
		{
			uint2 edge = source[offsetMem + (myBucket * DUCK_A_EDGES_64) + edgeIndex];

			int bucket = (edge.x >> 6) & (64 - 1);

			__syncthreads();

			int counter = min((int)atomicAdd(counters + bucket, 1), (int)15);

			tmp[bucket][counter] = edge;

			__syncthreads();

			int localIdx = min(16, counters[lid]);

			if (localIdx >= 8)
			{
				int newCount = (localIdx - 8);
				counters[lid] = newCount;

				{
					int cnt = min((int)atomicAdd(destinationIndexes + startBlock * 64 + myBucket * 64 + lid, 8), (int)(DUCK_A_EDGES - 8));

					{
						destination[((myBucket * 64 + lid) * DUCK_A_EDGES + cnt) / 4] = Pack4edges(tmp[lid][0], tmp[lid][1], tmp[lid][2], tmp[lid][3]);
						destination[((myBucket * 64 + lid) * DUCK_A_EDGES + cnt + 4) / 4] = Pack4edges(tmp[lid][4], tmp[lid][5], tmp[lid][6], tmp[lid][7]);
					}
				}

				for (int t = 0; t < newCount; t++)
				{
					tmp[lid][t] = tmp[lid][t + 8];
				}

			}
		}
	}

	__syncthreads();

	{
		int localIdx = min(16, counters[lid]);

		if (localIdx >= 4)
		{
			int cnt = min((int)atomicAdd(destinationIndexes + startBlock * 64 + myBucket * 64 + lid, 4), (int)(DUCK_A_EDGES - 4));
			destination[((myBucket * 64 + lid) * DUCK_A_EDGES + cnt) / 4] = Pack4edges(tmp[lid][0], tmp[lid][1], tmp[lid][2], tmp[lid][3]);
		}
		if (localIdx > 4)
		{
			int cnt = min((int)atomicAdd(destinationIndexes + startBlock * 64 + myBucket * 64 + lid, 4), (int)(DUCK_A_EDGES - 4));
			destination[((myBucket * 64 + lid) * DUCK_A_EDGES + cnt) / 4] = Pack4edges(
				tmp[lid][4],
				localIdx > 5 ? tmp[lid][5] : make_uint2(0, 0),
				localIdx > 6 ? tmp[lid][6] : make_uint2(0, 0),
				localIdx > 7 ? tmp[lid][7] : make_uint2(0, 0));
		}
	}
}

__device__ __forceinline__  void Increase2bCounter(u32 * ecounters, const int bucket)
{
	int word = bucket >> 5;
	unsigned char bit = bucket & 0x1F;
	u32 mask = 1 << bit;

	u32 old = atomicOr(ecounters + word, mask) & mask;

	if (old > 0)
		atomicOr(ecounters + word + 4096, mask);
}

__device__ __forceinline__  bool Read2bCounter(u32 * ecounters, const int bucket)
{
	int word = bucket >> 5;
	unsigned char bit = bucket & 0x1F;
	u32 mask = 1 << bit;

	return (ecounters[word + 4096] & mask) > 0;
}

template<int bktInSize, int bktOutSize>
__global__   void FluffyRound(const uint2 * source, uint2 * destination, const int * sourceIndexes, int * destinationIndexes)
{
	const int gid = blockDim.x * blockIdx.x + threadIdx.x;
	const int lid = threadIdx.x;
	const int group = blockIdx.x;

	__shared__ u32 ecounters[8192];

	const int edgesInBucket = min(sourceIndexes[group], bktInSize);
	const int loops = (edgesInBucket + CTHREADS) / CTHREADS;

	ecounters[lid] = 0;
	ecounters[lid + 1024] = 0;
	ecounters[lid + (1024 * 2)] = 0;
	ecounters[lid + (1024 * 3)] = 0;
	ecounters[lid + (1024 * 4)] = 0;
	ecounters[lid + (1024 * 5)] = 0;
	ecounters[lid + (1024 * 6)] = 0;
	ecounters[lid + (1024 * 7)] = 0;

	__syncthreads();

	for (int i = 0; i < loops; i++)
	{
		const int lindex = (i * CTHREADS) + lid;

		if (lindex < edgesInBucket)
		{
			const int index = (bktInSize * group) + lindex;

			uint2 edge = source[index];

			if (edge.x == 0 && edge.y == 0) continue;

			Increase2bCounter(ecounters, (edge.x & EDGEMASK) >> 12);
		}
	}

	__syncthreads();

	for (int i = 0; i < loops; i++)
	{
		const int lindex = (i * CTHREADS) + lid;

		if (lindex < edgesInBucket)
		{
			const int index = (bktInSize * group) + lindex;

			uint2 edge = source[index];

			if (edge.x == 0 && edge.y == 0) continue;

			if (Read2bCounter(ecounters, (edge.x & EDGEMASK) >> 12))
			{
				const int bucket = edge.y & BKTMASK4K;
				const int bktIdx = min(atomicAdd(destinationIndexes + bucket, 1), bktOutSize - 1);
				destination[(bucket * bktOutSize) + bktIdx] = make_uint2(edge.y, edge.x);
			}
		}
	}

}

template __global__ void FluffyRound<DUCK_A_EDGES, DUCK_B_EDGES>(const uint2 * source, uint2 * destination, const int * sourceIndexes, int * destinationIndexes);
template __global__ void FluffyRound<DUCK_B_EDGES, DUCK_B_EDGES>(const uint2 * source, uint2 * destination, const int * sourceIndexes, int * destinationIndexes);

__global__   void /*Magical*/FluffyTail/*Pony*/(const uint2 * source, uint2 * destination, const int * sourceIndexes, int * destinationIndexes)
{
	const int lid = threadIdx.x;
	const int group = blockIdx.x;

	int myEdges = sourceIndexes[group];
	__shared__ int destIdx;

	if (lid == 0)
		destIdx = atomicAdd(destinationIndexes, myEdges);

	__syncthreads();

	if (lid < myEdges)
	{
		destination[destIdx + lid] = source[group * DUCK_B_EDGES + lid];
	}
}

static u32 hostB[2 * 260000];
static u64 h_mydata[42];

int main()
{
	std::ofstream myfile;

	u32 * buffer = new u32[150000 * 2];

	const size_t bufferSize = DUCK_SIZE_A * 1024 * 4096 * 8;
	const size_t bufferSize2 = DUCK_SIZE_B * 1024 * 4096 * 8;
	const size_t indexesSize = 128 * 128 * 4;

	const unsigned int edges = (1 << 29);

	int * bufferA;
	int * bufferB;
	int * indexesE;
	int * indexesE2;

	u32 hostA[256 * 256];

	hipError_t cudaStatus;
	size_t free_device_mem = 0;
	size_t total_device_mem = 0;

	unsigned long long k0 = 0xa34c6a2bdaa03a14ULL;
	unsigned long long k1 = 0xd736650ae53eee9eULL;
	unsigned long long k2 = 0x9a22f05e3bffed5eULL;
	unsigned long long k3 = 0xb8d55478fa3a606dULL;

	unsigned long long nonce = 0;

#ifdef _WIN32
	HANDLE handle = CreateFileMappingW(INVALID_HANDLE_VALUE, NULL, PAGE_READWRITE, 0, 8000000, L"CuckoDataSend");
	u32 * sharedData = (u32*)MapViewOfFile(handle, FILE_MAP_READ | FILE_MAP_WRITE, 0, 0, 8000000);
#else

#endif


	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		goto Error;
	}

	hipMemGetInfo(&free_device_mem, &total_device_mem);

	fprintf(stderr, "Currently available amount of device memory: %zu bytes\n", free_device_mem);
	fprintf(stderr, "Total amount of device memory: %zu bytes\n", total_device_mem);

	cudaStatus = hipMalloc((void**)&bufferA, bufferSize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "status: %s\n", hipGetErrorString(cudaStatus));
		fprintf(stderr, "hipMalloc failed buffer A 4GB!\n");
		goto Error;
	}

	fprintf(stderr, "Allociating buffer 1\n");

	hipMemGetInfo(&free_device_mem, &total_device_mem);

	//printf("Buffer A: Currently available amount of device memory: %zu bytes\n", free_device_mem);

	fprintf(stderr, "Allociating buffer 2\n");

	cudaStatus = hipMalloc((void**)&bufferB, bufferSize2);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "status: %s\n", hipGetErrorString(cudaStatus));
		fprintf(stderr, "hipMalloc failed buffer B 3GB!\n");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&indexesE, indexesSize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed Index array 1!\n");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&indexesE2, indexesSize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed Index array 2!\n");
		goto Error;
	}

	hipMemGetInfo(&free_device_mem, &total_device_mem);

	fprintf(stderr, "Currently available amount of device memory: %zu bytes\n", free_device_mem);

	fprintf(stderr, "CUDA device armed\n");

	// loop starts here
	// wait for header hashes, nonce+r

	while (1)
	{
		fprintf(stderr, "#r\n"); // ready
								 // read commands from stdin

		while (getchar() != '#');
		int command = getchar();

		// parse command

		if (command == 'e')
		{
			// exit loop and terminate
			break;
		}
		else if (command == 't')
		{
			// comamnded to trim edges
			// parse k0 k1 k2 k3 nonce

			scanf("%llu %llu %llu %llu %llu", &k0, &k1, &k2, &k3, &nonce);
			fprintf(stderr, "#a\n"); // ack
			fprintf(stderr, "Trimming: %llx %llx %llx %llx\n", k0, k1, k2, k3); // ack
		}
		else if (command == 's')
		{
			scanf("%llu %llu %llu %llu %llu", &k0, &k1, &k2, &k3, &nonce);
			for (int i = 0; i < 42; i++)
				scanf(" %llu", &(h_mydata[i]));
			hipMemcpyToSymbol(HIP_SYMBOL(recovery), h_mydata, 42 * 8);
			hipDeviceSynchronize();

			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess)
				fprintf(stderr, "status symbol copy: %s\n", hipGetErrorString(cudaStatus));

			// recover solution
			hipMemset(indexesE2, 0, indexesSize);
			FluffyRecovery << < 512, 256 >> >(k0, k1, k2, k3, (ulonglong4 *)bufferA, (int *)indexesE2);
			hipDeviceSynchronize();
			hipMemcpy(hostA, indexesE2, 42 * 8, hipMemcpyDeviceToHost);

			fprintf(stderr, "#s"); 
			for (int i = 0; i < 42; i++)
				fprintf(stderr, " %lu", hostA[i]);
			fprintf(stderr, "\n");

			continue;
		}
		else
			continue;

		hipMemset(indexesE, 0, indexesSize);
		hipMemset(indexesE2, 0, indexesSize);

		hipDeviceSynchronize();

		

#ifdef VRAMSMALL
		FluffySeed2A << < 512, 64 >> > (k0, k1, k2, k3, (ulonglong4 *)bufferA, (int *)indexesE2);

		FluffySeed2B << < 32 * BKTGRAN, 64 >> > ((const uint2 *)bufferA, (ulonglong4 *)bufferB, (const int *)indexesE2, (int *)indexesE, 0);
		hipMemcpy(bufferA, bufferB, bufferSize / 2, hipMemcpyDeviceToDevice);

		FluffySeed2B << < 32 * BKTGRAN, 64 >> > ((const uint2 *)bufferA, (ulonglong4 *)bufferB, (const int *)indexesE2, (int *)indexesE, 32);
		cudaStatus = hipMemcpy(&((char *)bufferA)[bufferSize / 2], bufferB, bufferSize / 2, hipMemcpyDeviceToDevice);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess)
			fprintf(stderr, "status memcpy: %s\n", hipGetErrorString(cudaStatus));


		hipMemset(indexesE2, 0, indexesSize);
		FluffyRound<DUCK_A_EDGES, DUCK_B_EDGES> << < 4096, 1024 >> > ((const uint2 *)bufferA, (uint2 *)bufferB, (const int *)indexesE, (int *)indexesE2);

#else
		FluffySeed2A << < 512, 64 >> > (k0, k1, k2, k3, (ulonglong4 *)bufferA, (int *)indexesE);

		FluffySeed2B << < 32 * BKTGRAN, 64 >> > ((const uint2 *)bufferA, (ulonglong4 *)bufferB, (const int *)indexesE, (int *)indexesE2, 0);
		FluffySeed2B << < 32 * BKTGRAN, 64 >> > ((const uint2 *)bufferA, (ulonglong4 *)bufferB, (const int *)indexesE, (int *)indexesE2, 32);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess)
			fprintf(stderr, "status memcpy: %s\n", hipGetErrorString(cudaStatus));

		hipMemset(indexesE, 0, indexesSize);
		FluffyRound<DUCK_A_EDGES, DUCK_B_EDGES> << < 4096, 1024 >> > ((const uint2 *)bufferB, (uint2 *)bufferA, (const int *)indexesE2, (int *)indexesE);
		hipMemset(indexesE2, 0, indexesSize);
		FluffyRound<DUCK_B_EDGES, DUCK_B_EDGES> << < 4096, 1024 >> > ((const uint2 *)bufferA, (uint2 *)bufferB, (const int *)indexesE, (int *)indexesE2);
#endif

		hipDeviceSynchronize();

		for (int i = 0; i < 80; i++)
		{
			hipMemset(indexesE, 0, indexesSize);
			FluffyRound<DUCK_B_EDGES, DUCK_B_EDGES> << < 4096, 1024 >> > ((const uint2 *)bufferB, (uint2 *)bufferA, (const int *)indexesE2, (int *)indexesE);
			hipMemset(indexesE2, 0, indexesSize);
			FluffyRound<DUCK_B_EDGES, DUCK_B_EDGES> << < 4096, 1024 >> > ((const uint2 *)bufferA, (uint2 *)bufferB, (const int *)indexesE, (int *)indexesE2);
			hipMemset(indexesE, 0, indexesSize);
		}

		hipDeviceSynchronize();

		FluffyTail << < 4096, 1024 >> > ((const uint2 *)bufferB, (uint2 *)bufferA, (const int *)indexesE2, (int *)indexesE);
		hipMemcpy(hostA, indexesE, 64 * 64 * 4, hipMemcpyDeviceToHost);

		int pos = hostA[0];
		if (pos > 0 && pos < 500000)
			hipMemcpy(&((u64 *)buffer)[0], &((u64 *)bufferA)[0], pos * 8, hipMemcpyDeviceToHost);

		hipDeviceSynchronize();

		fprintf(stderr, "Trimmed to: %d edges\n", pos);

#ifdef _WIN32
		if (sharedData != NULL)
		{
			sharedData[0] = pos;
			for (int i = 0; i < pos; i++)
			{
				sharedData[i * 2 + 0 + 1] = buffer[i * 2 + 0];
				sharedData[i * 2 + 1 + 1] = buffer[i * 2 + 1];
			}
		}
		else
		{
			fprintf(stderr, "Memory mapped file write error!\n");
			goto Error;
		}
#else
		{
			auto myfile = std::fstream("edges/data.bin", std::ios::out | std::ios::binary);
			myfile.write((const char *)&pos, 4);
			myfile.write((const char *)buffer, pos * 8);
			myfile.close();
		}
#endif
		fprintf(stderr, "#e %d \n", pos);


	}

	delete buffer;
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
		fprintf(stderr, "status: %s\n", hipGetErrorString(cudaStatus));


Error:
#ifdef _WIN32
	if (handle != NULL)
		CloseHandle(handle);
#else
#endif

	fprintf(stderr, "CUDA terminating...\n");
	fprintf(stderr, "#x\n");
	hipFree(bufferA);
	hipFree(bufferB);
	hipFree(indexesE);
	hipFree(indexesE2);
	hipDeviceReset();
	return 0;
}
